#include <iostream>
#include <hip/hip_runtime.h>

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

void checkCudaError(hipError_t error, const char *function) {
    if (error != hipSuccess) {
        std::cerr << "Error in " << function << ": " << hipGetErrorString(error) << std::endl;
        exit(1);
    }
}

int main() {
    int c;
    int *dev_c;

    // Get and print the CUDA Runtime version
    int runtimeVer = 0;
    hipRuntimeGetVersion(&runtimeVer);
    std::cout << "CUDA Runtime Version: " << runtimeVer / 1000 << "." << (runtimeVer % 100) / 10 << std::endl;

    // Allocate memory on the GPU
    checkCudaError(hipMalloc((void**)&dev_c, sizeof(int)), "hipMalloc");

    // Launch the add kernel on the GPU
    add<<<1,1>>>(2, 7, dev_c);
    checkCudaError(hipGetLastError(), "Kernel launch");

    // Copy the result back to the host
    checkCudaError(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost), "hipMemcpy");

    std::cout << "2 + 7 = " << c << std::endl;

    // Free the memory allocated on the GPU
    hipFree(dev_c);
    
    return 0;
}
